#include <iostream>
#include <cstdlib>
#include <cassert>
#include <zlib.h>
#include <png.h>
#include <hip/hip_runtime.h>

#define MASK_N 2
#define MASK_X 5
#define MASK_Y 5
#define SCALE 8

// Device mask
__constant__ int mask[MASK_N][MASK_X][MASK_Y] = { 
    {{ -1, -4, -6, -4, -1},
     { -2, -8,-12, -8, -2},
     {  0,  0,  0,  0,  0}, 
     {  2,  8, 12,  8,  2}, 
     {  1,  4,  6,  4,  1}},
    {{ -1, -2,  0,  2,  1}, 
     { -4, -8,  0,  8,  4}, 
     { -6,-12,  0, 12,  6}, 
     { -4, -8,  0,  8,  4}, 
     { -1, -2,  0,  2,  1}} 
};

int read_png(const char* filename, unsigned char** image, unsigned* height, 
             unsigned* width, unsigned* channels) {

    unsigned char sig[8];
    FILE* infile;
    infile = fopen(filename, "rb");

    fread(sig, 1, 8, infile);
    if (!png_check_sig(sig, 8))
        return 1;   /* bad signature */

    png_structp png_ptr;
    png_infop info_ptr;

    png_ptr = png_create_read_struct(PNG_LIBPNG_VER_STRING, NULL, NULL, NULL);
    if (!png_ptr)
        return 4;   /* out of memory */
  
    info_ptr = png_create_info_struct(png_ptr);
    if (!info_ptr) {
        png_destroy_read_struct(&png_ptr, NULL, NULL);
        return 4;   /* out of memory */
    }

    png_init_io(png_ptr, infile);
    png_set_sig_bytes(png_ptr, 8);
    png_read_info(png_ptr, info_ptr);
    int bit_depth, color_type;
    png_get_IHDR(png_ptr, info_ptr, width, height, &bit_depth, &color_type, NULL, NULL, NULL);

    png_uint_32  i, rowbytes;
    png_bytep  row_pointers[*height];
    png_read_update_info(png_ptr, info_ptr);
    rowbytes = png_get_rowbytes(png_ptr, info_ptr);
    *channels = (int) png_get_channels(png_ptr, info_ptr);

    if ((*image = (unsigned char *) malloc(rowbytes * *height)) == NULL) {
        png_destroy_read_struct(&png_ptr, &info_ptr, NULL);
        return 3;
    }

    for (i = 0;  i < *height;  ++i)
        row_pointers[i] = *image + i * rowbytes;
    png_read_image(png_ptr, row_pointers);
    png_read_end(png_ptr, NULL);
    return 0;
}

void write_png(const char* filename, png_bytep image, const unsigned height, const unsigned width, 
               const unsigned channels) {
    FILE* fp = fopen(filename, "wb");
    png_structp png_ptr = png_create_write_struct(PNG_LIBPNG_VER_STRING, NULL, NULL, NULL);
    png_infop info_ptr = png_create_info_struct(png_ptr);
    png_init_io(png_ptr, fp);
    png_set_IHDR(png_ptr, info_ptr, width, height, 8,
                 PNG_COLOR_TYPE_RGB, PNG_INTERLACE_NONE,
                 PNG_COMPRESSION_TYPE_DEFAULT, PNG_FILTER_TYPE_DEFAULT);
    png_set_filter(png_ptr, 0, PNG_NO_FILTERS);
    png_write_info(png_ptr, info_ptr);
    png_set_compression_level(png_ptr, 1);

    png_bytep row_ptr[height];
    for (int i = 0; i < height; ++ i) {
        row_ptr[i] = image + i * width * channels * sizeof(unsigned char);
    }
    png_write_image(png_ptr, row_ptr);
    png_write_end(png_ptr, NULL);
    png_destroy_write_struct(&png_ptr, &info_ptr);
    fclose(fp);
}


// Kernel function for Sobel filter
__global__ void sobelKernel(unsigned char* s, unsigned char* t, unsigned height, unsigned width, unsigned channels) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x >= width || y >= height) return;

    double val[MASK_N * 3] = {0.0};
    int xBound = MASK_X / 2;
    int yBound = MASK_Y / 2;

    for (int i = 0; i < MASK_N; ++i) {
        for (int v = -yBound; v <= yBound; ++v) {
            for (int u = -xBound; u <= xBound; ++u) {
                int nx = x + u;
                int ny = y + v;
                
                if (nx >= 0 && nx < width && ny >= 0 && ny < height) {
                    int R = s[channels * (width * ny + nx) + 2];
                    int G = s[channels * (width * ny + nx) + 1];
                    int B = s[channels * (width * ny + nx)];

                    val[i * 3 + 2] += R * mask[i][u + xBound][v + yBound];
                    val[i * 3 + 1] += G * mask[i][u + xBound][v + yBound];
                    val[i * 3] += B * mask[i][u + xBound][v + yBound];
                }
            }
        }
    }

    double totalR = 0.0, totalG = 0.0, totalB = 0.0;
    for (int i = 0; i < MASK_N; ++i) {
        totalR += val[i * 3 + 2] * val[i * 3 + 2];
        totalG += val[i * 3 + 1] * val[i * 3 + 1];
        totalB += val[i * 3] * val[i * 3];
    }

    totalR = sqrt(totalR) / SCALE;
    totalG = sqrt(totalG) / SCALE;
    totalB = sqrt(totalB) / SCALE;

    t[channels * (width * y + x) + 2] = min(255.0, totalR);
    t[channels * (width * y + x) + 1] = min(255.0, totalG);
    t[channels * (width * y + x)] = min(255.0, totalB);
}

int main(int argc, char** argv) {
    assert(argc == 3);
    unsigned height, width, channels;
    unsigned char* host_s = NULL;
    read_png(argv[1], &host_s, &height, &width, &channels);
    unsigned char* host_t = (unsigned char*) malloc(height * width * channels * sizeof(unsigned char));

    // Allocate device memory
    unsigned char *dev_s, *dev_t;
    hipMalloc((void**)&dev_s, height * width * channels * sizeof(unsigned char));
    hipMalloc((void**)&dev_t, height * width * channels * sizeof(unsigned char));

    // Copy input image to device
    hipMemcpy(dev_s, host_s, height * width * channels * sizeof(unsigned char), hipMemcpyHostToDevice);

    // Launch kernel
    dim3 block(16, 16);
    dim3 grid((width + block.x - 1) / block.x, (height + block.y - 1) / block.y);
    sobelKernel<<<grid, block>>>(dev_s, dev_t, height, width, channels);
    hipDeviceSynchronize();

    // Copy result back to host
    hipMemcpy(host_t, dev_t, height * width * channels * sizeof(unsigned char), hipMemcpyDeviceToHost);

    write_png(argv[2], host_t, height, width, channels);

    // Free device memory
    hipFree(dev_s);
    hipFree(dev_t);
    free(host_s);
    free(host_t);

    return 0;
}
